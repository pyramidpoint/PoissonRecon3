#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <random>
#include <time.h>
#include <math.h>


__global__ void marching_cubes_filter(int *lookup_two, int *mcIsLeaf, float * mcCubeValue, float *mcCubePosition, int sizeSolution, float *triangles, float Isovalue) {
	//printf("hello world\n");
	//const int global_x = blockIdx.x * blockDim.x + threadIdx.x;
	//const int global_y = blockIdx.y * blockDim.y + threadIdx.y;
	//const int global_z = blockIdx.z * blockDim.z + threadIdx.z;

	//const int node = global_z * data_width * data_height + global_y * data_width + global_x;

	//if(global_x + 1 < data_width-1 && global_y + 1 < data_height-1  && global_z + 1 < data_depth-1){
	int node = blockIdx.x * blockDim.x + threadIdx.x;

	if (node < sizeSolution)
	{

		if (mcIsLeaf[node] == 1)
		{
			//printf("node is %d\n", node);
			for (unsigned int tm = 0; tm<(5 * 3 * 3); tm++) {
				triangles[node* (5 * 3 * 3) + tm] = 0.0;
			}

			// double check that these refer to the right vertices
			float x = mcCubePosition[4 * node + 0];
			float y = mcCubePosition[4 * node + 1];
			float z = mcCubePosition[4 * node + 2];
			float width = mcCubePosition[4 * node + 3];
			float cube[8][3]{
				{ x, y, z },//0 0 0
				{ x + width, y,z },//1 0 0
				{ x + width,y + width, z },//1 1 0
				{ x , y + width, z },//0 1 0
				{ x, y, z + width },//0 0 1
				{ x, y + width, z + width },//0 1 1
				{ x + width, y + width, z + width },//1 1 1
				{ x + width, y, z + width } };//1 0 1
											  //edgeidx=i | (j << 1)) | (orientation << 2)  o j i
											  //o j i    x y z		 cube
											  //0 0 0(x) 0 0 0->1 0 0  0->1
											  //0 0 1    0 1 0->1 1 0  3->2
											  //0 1 0    0 0 1->1 0 1  4->7
											  //0 1 1    0 1 1->1 1 1  5->6
											  //1 0 0(y) 0 0 0->0 1 0  0->3
											  //1 0 1    1 0 0->1 1 0  1->2
											  //1 1 0    0 0 1->0 1 1  4->5
											  //1 1 1    1 0 1->1 1 1  7->6
											  //2 0 0(z) 0 0 0->0 0 1  0->4
											  //2 0 1    1 0 0->1 0 1  1->7
											  //2 1 0    0 1 0->0 1 1  3->5
											  //2 1 1    1 1 0->1 1 1  2->6

			int cubeindex[8] = { 1,2,8,4,16,32,128,64 };
			//���ݶ���ֵ�õ�����
			//printf("position is %f %f %f\n", x, y, z);
			int case_lookup_idx = 0;
			for (unsigned int ci = 0; ci<8; ci++) {
				//const int x = cube[ci][0];
				//const int y = cube[ci][1];
				//const int z = cube[ci][2];


				//const int vertex = z * data_width * data_height + y * data_width + x;
				/*
				if (v[Cube::CornerIndex(0,0,0)] < iso) idx |=   1;
				if (v[Cube::CornerIndex(1,0,0)] < iso) idx |=   2;
				if (v[Cube::CornerIndex(1,1,0)] < iso) idx |=   4;
				if (v[Cube::CornerIndex(0,1,0)] < iso) idx |=   8;
				if (v[Cube::CornerIndex(0,0,1)] < iso) idx |=  16;
				if (v[Cube::CornerIndex(1,0,1)] < iso) idx |=  32;
				if (v[Cube::CornerIndex(1,1,1)] < iso) idx |=  64;
				if (v[Cube::CornerIndex(0,1,1)] < iso) idx |= 128;
				*/
				if (mcCubeValue[node * 8 + ci] <= Isovalue) {
					case_lookup_idx |= cubeindex[ci];
				}

			}
			for (int i = 0; i < 8; i++)
			{
				//printf("%f ", mcCubeValue[node * 8 + i]);
			}
			//printf("%f %f %f %f %f %f %f %f %d %d\n", mcCubeValue[node * 8 + 0], mcCubeValue[node * 8 + 1], mcCubeValue[node * 8 + 2], mcCubeValue[node * 8 + 3], mcCubeValue[node * 8 + 4], mcCubeValue[node * 8 + 5], mcCubeValue[node * 8 + 6], mcCubeValue[node * 8 + 7], case_lookup_idx,node);
			//���������õ��߼���
			float edge_actual[12][6] = {

				{ cube[0][0],cube[0][1],cube[0][2],cube[1][0],cube[1][1],cube[1][2] },
				{ cube[3][0],cube[3][1],cube[3][2],cube[2][0],cube[2][1],cube[2][2] },
				{ cube[4][0],cube[4][1],cube[4][2],cube[7][0],cube[7][1],cube[7][2] },
				{ cube[5][0],cube[5][1],cube[5][2],cube[6][0],cube[6][1],cube[6][2] },

				{ cube[0][0],cube[0][1],cube[0][2],cube[3][0],cube[3][1],cube[3][2] },
				{ cube[1][0],cube[1][1],cube[1][2],cube[2][0],cube[2][1],cube[2][2] },
				{ cube[4][0],cube[4][1],cube[4][2],cube[5][0],cube[5][1],cube[5][2] },
				{ cube[7][0],cube[7][1],cube[7][2],cube[6][0],cube[6][1],cube[6][2] },

				{ cube[0][0],cube[0][1],cube[0][2],cube[4][0],cube[4][1],cube[4][2] },
				{ cube[1][0],cube[1][1],cube[1][2],cube[7][0],cube[7][1],cube[7][2] },
				{ cube[3][0],cube[3][1],cube[3][2],cube[5][0],cube[5][1],cube[5][2] },
				{ cube[2][0],cube[2][1],cube[2][2],cube[6][0],cube[6][1],cube[6][2] }

			};
			//edgeidx=i | (j << 1)) | (orientation << 2)  o j i
			//o j i    x y z		 cube
			//0 0 0(x) 0 0 0->1 0 0  0->1
			//0 0 1    0 1 0->1 1 0  3->2
			//0 1 0    0 0 1->1 0 1  4->7
			//0 1 1    0 1 1->1 1 1  5->6
			//1 0 0(y) 0 0 0->0 1 0  0->3
			//1 0 1    1 0 0->1 1 0  1->2
			//1 1 0    0 0 1->0 1 1  4->5
			//1 1 1    1 0 1->1 1 1  7->6
			//2 0 0(z) 0 0 0->0 0 1  0->4
			//2 0 1    1 0 0->1 0 1  1->7
			//2 1 0    0 1 0->0 1 1  3->5
			//2 1 1    1 1 0->1 1 1  2->6
			int edgecorner[12][6] = {
				{ 0,0,0,1,0,0 },
				{ 0,1,0,1,1,0 },
				{ 0,0,1,1,0,1 },
				{ 0,1,1,1,1,1 },
				{ 0,0,0,0,1,0 },
				{ 1,0,0,1,1,0 },
				{ 0,0,1,0,1,1 },
				{ 1,0,1,1,1,1 },
				{ 0,0,0,0,0,1 },
				{ 1,0,0,1,0,1 },
				{ 0,1,0,0,1,1 },
				{ 1,1,0,1,1,1 }
			};

			//printf("%d corresponding %d\n", node, case_lookup_idx);
			if (case_lookup_idx != 255 && case_lookup_idx != 0) {
				int current = 0;
				int edge_counter = 0;
				//printf("case_lookup_idx is %d\n", case_lookup_idx);
				for (int w = 0; w<16; w++) {
					current = lookup_two[case_lookup_idx * 16 + w];
					// current now gives an edge index so we need to add the point to the triangle list

					if (current != -1) {
						//printf("current!=1\n");
						float point1_x = edge_actual[current][0];
						float point1_y = edge_actual[current][1];
						float point1_z = edge_actual[current][2];
						int x_1 = edgecorner[current][0];
						int y_1 = edgecorner[current][1];
						int z_1 = edgecorner[current][2];
						int x_2 = edgecorner[current][3];
						int y_2 = edgecorner[current][4];
						int z_2 = edgecorner[current][5];
						float point2_x = edge_actual[current][3];
						float point2_y = edge_actual[current][4];
						float point2_z = edge_actual[current][5];
						int orientation = current >> 2;
						//iso-x0/x1-iso=1-x/x
						//x1-iso-x1x+siox=isox+xx0
						double averageRoot;
						//printf("x1 = %d %d %d\n", x_1, y_1, z_1);
						//printf("x2 = %d %d %d\n", x_2, y_2, z_2);
						//printf("x1=%d\n", (z_1 << 2) + (y_1 << 1) + x_1);
						//printf("x2=%d\n", (z_2 << 2) + (y_2 << 1) + x_2);
						//printf("point1_x is %f %f %f\n", point1_x, point1_y, point1_z);
						//printf("point2_x is %f %f %f\n", point2_x, point2_y, point2_z);
						//printf("%f %f %f\n", (point2_x - point1_x), (point2_y - point1_y), (point2_z - point1_z));
						averageRoot = (Isovalue - mcCubeValue[node * 8 + (z_1 << 2) + (y_1 << 1) + x_1]) / (mcCubeValue[node * 8 + (z_2 << 2) + (y_2 << 1) + x_2] - mcCubeValue[node * 8 + (z_1 << 2) + (y_1 << 1) + x_1]);
						//	printf("%f\n", averageRoot);
						//printf("Isovalue - mcCubeValue[%d*8+(z_1 << 2) + (y_1 << 1) + x_1] is %f\n",node, Isovalue - mcCubeValue[node * 8 + (z_1 << 2) + (y_1 << 1) + x_1]);
						//printf(" (mcCubeValue[%d*8+(z_2 << 2) + (y_2 << 1) + x_2]: %f - mcCubeValue[*8+(z_1 << 2) + (y_1 << 1) + x_1] )%f  is %f\n",node, mcCubeValue[node *8 + (z_2 << 2) + (y_2 << 1) + x_2], mcCubeValue[node * 8 + (z_1 << 2) + (y_1 << 1) + x_1],(mcCubeValue[node * 8 + (z_2 << 2) + (y_2 << 1) + x_2] - mcCubeValue[node * 8 + (z_1 << 2) + (y_1 << 1) + x_1]));
						//	printf("averageRoot is %f\n", averageRoot);
						if (orientation == 0)
						{
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 0] = (((float)point1_x + (float)width*averageRoot));
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 1] = (((float)point1_y + (float)point2_y) / 2.0);
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 2] = (((float)point1_z + (float)point2_z) / 2.0);// could do better interpolation here
						}
						else if (orientation == 1)
						{
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 0] = (((float)point1_x + (float)point2_x) / 2.0);
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 1] = (((float)point1_y + (float)width*averageRoot));
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 2] = (((float)point1_z + (float)point2_z) / 2.0);// could do better interpolation here
						}
						else if (orientation == 2)
						{
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 0] = (((float)point1_x + (float)point2_x) / 2.0);
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 1] = (((float)point1_y + (float)point2_y) / 2.0);
							triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 2] = (((float)point1_z + (float)width*averageRoot));// could do better interpolation here
						}
						//printf("%f %f %f  ", triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 0], triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 1], triangles[node * (5 * 3 * 3) + (edge_counter * 3) + 2]);




						edge_counter++;
					}



				}
				//printf("\n");
			}
		}
	}

}






extern "C"
hipError_t MC(float * mcCubeValue, int * mcIsLeaf, float *mcCubePosition, int sizeSolution, float *trangles, float Isovalue)
{
	hipFree(0);
	float *dev_mcCubeValue = NULL;
	float *dev_mcCubePosition = NULL;
	float *dev_trangles = NULL;
	int * dev_mcIsLeaf = NULL;
	int *device_lookup = NULL;

	hipError_t cudaStatus;
	//printf("welcome to mulWithCudamatrixn\n");
	cudaStatus = hipSetDevice(0);
	//printf("welcome to mulWithCudamatrixn\n");
	if (cudaStatus != hipSuccess)
	{
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	//else
	//{
	//	printf("CUDA-capable GPU has installed\n");
	//}


	cudaStatus = hipMalloc((void **)&dev_mcCubeValue, sizeSolution * 8 * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev_mcCubeValue failed!\n");
		goto Error;
	}
	//else
	//{
	//	printf("dev_dev_mcCubeValue has cudaMalloced\n");
	//}

	cudaStatus = hipMalloc((void **)&dev_mcCubePosition, sizeSolution * 4 * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev_mcCubePosition failed!\n");
		goto Error;
	}
	//else
	//{
	//	printf("dev_mcCubePosition has cudaMalloced\n");
	//}

	cudaStatus = hipMalloc((void **)&dev_trangles, sizeSolution * 5 * 3 * 3 * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev_trangles failed!\n");
		goto Error;
	}
	//else
	//{
	//	printf("dev_trangles has cudaMalloced\n");
	//}

	cudaStatus = hipMalloc((void **)&dev_mcIsLeaf, sizeSolution * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev_mcIsLeaf failed!\n");
		goto Error;
	}
	//else
	//{
	//	printf("dev_mcIsLeaf has cudaMalloced\n");
	//}



	cudaStatus = hipMemcpy(dev_mcCubeValue, mcCubeValue, sizeSolution * 8 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("cudamemcpy dev_mcCubeValue failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mcCubePosition, mcCubePosition, sizeSolution * 4 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy dev_mcCubePosition failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mcIsLeaf, mcIsLeaf, sizeSolution * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy dev_mcIsLeaf failed!\n");
		goto Error;
	}
	int triTable[256][16] =
	{
		{ -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,   5,   8,   5,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,   1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,  11,   1,   9,   1,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,  11,   8,  11,   1,   8,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   0,  10,   0,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   9,  10,   9,   5,  10,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,  10,   4,  11,   4,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,  10,   8,  11,   8,   0,  11,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,  11,  10,   9,  10,   4,   9,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,  11,   8,  11,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,   2,   8,   5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   4,   6,   9,   5,   6,   2,   9,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   5,  11,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   5,  11,   6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,   2,   8,   9,  11,   1,   9,   1,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,  11,   2,   2,  11,   1,   2,   1,   6,   6,   1,   4,  -1,  -1,  -1,  -1 },
		{ 1,  10,   4,   2,   8,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   0,   1,   6,   2,   1,  10,   6,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   4,   1,  10,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   2,   9,   5,   6,   2,   5,   1,   6,   1,  10,   6,  -1,  -1,  -1,  -1 },
		{ 2,   8,   6,   4,   5,  11,   4,  11,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   2,   0,   6,   2,   5,  11,   6,   5,  10,   6,  11,  -1,  -1,  -1,  -1 },
		{ 9,  11,  10,   9,  10,   4,   9,   4,   0,   8,   6,   2,  -1,  -1,  -1,  -1 },
		{ 9,  11,   2,   2,  11,   6,  10,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   9,   2,   4,   8,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   2,   7,   0,   7,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   5,   4,   2,   7,   4,   8,   2,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   9,   2,   5,  11,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   5,  11,   0,   4,   8,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   0,   2,   1,   2,   7,   1,   7,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   7,  11,   1,   2,   7,   1,   4,   2,   4,   8,   2,  -1,  -1,  -1,  -1 },
		{ 4,   1,  10,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   9,   2,   0,   1,  10,   0,  10,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   1,  10,   2,   7,   5,   0,   2,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,  10,   8,   1,  10,   2,   7,   1,   2,   5,   1,   7,  -1,  -1,  -1,  -1 },
		{ 7,   9,   2,  10,   4,   5,  11,  10,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,  10,   8,  11,   8,   0,  11,   0,   5,   9,   2,   7,  -1,  -1,  -1,  -1 },
		{ 11,  10,   7,   7,  10,   4,   7,   4,   2,   2,   4,   0,  -1,  -1,  -1,  -1 },
		{ 11,  10,   7,   7,  10,   2,   8,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   9,   8,   6,   7,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   6,   7,   0,   4,   7,   9,   0,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,   7,   5,   8,   6,   5,   0,   8,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   6,   7,   5,   4,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,  11,   1,   8,   6,   7,   9,   8,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   6,   7,   0,   4,   7,   9,   0,   7,  11,   1,   5,  -1,  -1,  -1,  -1 },
		{ 8,   1,   0,  11,   1,   8,   6,  11,   8,   7,  11,   6,  -1,  -1,  -1,  -1 },
		{ 11,   6,   7,   1,   6,  11,   6,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,  10,   4,   6,   7,   9,   6,   9,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   1,   9,   9,   1,  10,   9,  10,   7,   7,  10,   6,  -1,  -1,  -1,  -1 },
		{ 6,   7,   5,   8,   6,   5,   0,   8,   5,   1,  10,   4,  -1,  -1,  -1,  -1 },
		{ 1,   7,   5,  10,   7,   1,   7,  10,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,  10,   4,  11,   4,   5,   7,   9,   8,   6,   7,   8,  -1,  -1,  -1,  -1 },
		{ 0,   6,   9,   9,   6,   7,   6,   0,   5,   5,  11,  10,   5,  10,   6,  -1 },
		{ 8,   7,   0,   6,   7,   8,   4,   0,   7,  11,  10,   4,   7,  11,   4,  -1 },
		{ 11,  10,   6,  11,   6,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   5,   0,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,   4,   8,   9,   5,   4,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   1,   5,   3,   5,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,   7,   3,   1,   5,   7,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   1,   0,   3,   0,   9,   3,   9,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   8,   9,   4,   8,   7,   3,   4,   7,   1,   4,   3,  -1,  -1,  -1,  -1 },
		{ 1,  10,   4,   3,  11,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,  11,   7,   8,   0,   1,  10,   8,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   1,  10,   5,   0,   9,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   9,  10,   9,   5,  10,   5,   1,  11,   7,   3,  -1,  -1,  -1,  -1 },
		{ 4,   5,   7,   4,   7,   3,   4,   3,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   3,   3,   8,   0,   3,   0,   7,   7,   0,   5,  -1,  -1,  -1,  -1 },
		{ 4,   3,  10,   4,   7,   3,   4,   0,   7,   0,   9,   7,  -1,  -1,  -1,  -1 },
		{ 10,   8,   3,   3,   8,   7,   9,   7,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,   2,   0,   4,   2,   4,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,   8,   6,   2,   5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   4,   6,   9,   5,   6,   2,   9,   6,   3,  11,   7,  -1,  -1,  -1,  -1 },
		{ 8,   6,   2,   3,   1,   5,   3,   5,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   1,   5,   3,   5,   7,   6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1 },
		{ 3,   1,   0,   3,   0,   9,   3,   9,   7,   2,   8,   6,  -1,  -1,  -1,  -1 },
		{ 9,   4,   2,   2,   4,   6,   4,   9,   7,   7,   3,   1,   7,   1,   4,  -1 },
		{ 8,   6,   2,  11,   7,   3,   4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   0,   1,   6,   2,   1,  10,   6,   1,  11,   7,   3,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   4,   1,  10,   8,   6,   2,  11,   7,   3,  -1,  -1,  -1,  -1 },
		{ 11,   7,   3,   5,   2,   9,   5,   6,   2,   5,   1,   6,   1,  10,   6,  -1 },
		{ 4,   5,   7,   4,   7,   3,   4,   3,  10,   6,   2,   8,  -1,  -1,  -1,  -1 },
		{ 10,   5,   3,   3,   5,   7,   5,  10,   6,   6,   2,   0,   6,   0,   5,  -1 },
		{ 8,   6,   2,   4,   3,  10,   4,   7,   3,   4,   0,   7,   0,   9,   7,  -1 },
		{ 9,   7,  10,  10,   7,   3,  10,   6,   9,   6,   2,   9,  -1,  -1,  -1,  -1 },
		{ 3,  11,   9,   2,   3,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   8,   0,   2,   3,  11,   2,  11,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   2,   3,   0,   3,  11,   0,  11,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   3,   8,   8,   3,  11,   8,  11,   4,   4,  11,   5,  -1,  -1,  -1,  -1 },
		{ 2,   3,   1,   2,   1,   5,   2,   5,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   3,   1,   2,   1,   5,   2,   5,   9,   0,   4,   8,  -1,  -1,  -1,  -1 },
		{ 0,   2,   3,   0,   3,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   3,   8,   8,   3,   4,   1,   4,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,  10,   4,   9,   2,   3,  11,   9,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   0,  10,   0,   1,   3,  11,   9,   2,   3,   9,  -1,  -1,  -1,  -1 },
		{ 0,   2,   3,   0,   3,  11,   0,  11,   5,   1,  10,   4,  -1,  -1,  -1,  -1 },
		{ 5,   2,  11,  11,   2,   3,   2,   5,   1,   1,  10,   8,   1,   8,   2,  -1 },
		{ 10,   2,   3,   9,   2,  10,   4,   9,  10,   5,   9,   4,  -1,  -1,  -1,  -1 },
		{ 5,  10,   0,   0,  10,   8,  10,   5,   9,   9,   2,   3,   9,   3,  10,  -1 },
		{ 0,   2,   4,   4,   2,  10,   3,  10,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   8,   2,  10,   2,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   9,   8,   3,  11,   8,   6,   3,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,  11,   9,   3,  11,   0,   4,   3,   0,   6,   3,   4,  -1,  -1,  -1,  -1 },
		{ 11,   5,   3,   5,   0,   3,   0,   6,   3,   0,   8,   6,  -1,  -1,  -1,  -1 },
		{ 3,   4,   6,  11,   4,   3,   4,  11,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   1,   6,   6,   1,   5,   6,   5,   8,   8,   5,   9,  -1,  -1,  -1,  -1 },
		{ 0,   6,   9,   4,   6,   0,   5,   9,   6,   3,   1,   5,   6,   3,   5,  -1 },
		{ 3,   1,   6,   6,   1,   8,   0,   8,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   1,   4,   3,   4,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   9,   8,   3,  11,   8,   6,   3,   8,   4,   1,  10,  -1,  -1,  -1,  -1 },
		{ 3,   9,   6,  11,   9,   3,  10,   6,   9,   0,   1,  10,   9,   0,  10,  -1 },
		{ 4,   1,  10,  11,   5,   3,   5,   0,   3,   0,   6,   3,   0,   8,   6,  -1 },
		{ 5,  10,   6,   1,  10,   5,   6,  11,   5,   6,   3,  11,  -1,  -1,  -1,  -1 },
		{ 10,   5,   3,   4,   5,  10,   6,   3,   5,   9,   8,   6,   5,   9,   6,  -1 },
		{ 6,   3,  10,   9,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,  10,   0,   0,  10,   4,   0,   8,   3,   8,   6,   3,  -1,  -1,  -1,  -1 },
		{ 6,   3,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   6,  10,   0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   6,  10,   8,   9,   5,   8,   5,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   1,   5,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,   1,   5,  11,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   0,   9,  11,   1,   0,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,  11,   8,  11,   1,   8,   1,   4,  10,   3,   6,  -1,  -1,  -1,  -1 },
		{ 4,   1,   3,   6,   4,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   1,   3,   8,   0,   3,   6,   8,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   3,   6,   4,   1,   3,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,   6,   6,   9,   5,   6,   5,   3,   3,   5,   1,  -1,  -1,  -1,  -1 },
		{ 6,   4,   5,   6,   5,  11,   6,  11,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   6,   8,   0,   3,   6,   0,   5,   3,   5,  11,   3,  -1,  -1,  -1,  -1 },
		{ 3,   9,  11,   0,   9,   3,   6,   0,   3,   4,   0,   6,  -1,  -1,  -1,  -1 },
		{ 8,   9,   6,   6,   9,   3,  11,   3,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   2,   0,  10,   3,   0,   4,  10,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   8,  10,   3,   8,   3,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   3,   2,  10,   3,   9,   5,  10,   9,   4,  10,   5,  -1,  -1,  -1,  -1 },
		{ 11,   1,   5,   2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   2,   0,  10,   3,   0,   4,  10,   0,   5,  11,   1,  -1,  -1,  -1,  -1 },
		{ 9,  11,   1,   9,   1,   0,   2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1 },
		{ 10,   2,   4,   3,   2,  10,   1,   4,   2,   9,  11,   1,   2,   9,   1,  -1 },
		{ 1,   3,   2,   4,   1,   2,   8,   4,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   1,   3,   2,   0,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   3,   2,   4,   1,   2,   8,   4,   2,   9,   5,   0,  -1,  -1,  -1,  -1 },
		{ 9,   3,   2,   5,   3,   9,   3,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   2,  11,  11,   2,   8,  11,   8,   5,   5,   8,   4,  -1,  -1,  -1,  -1 },
		{ 5,   2,   0,  11,   2,   5,   2,  11,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   3,   8,   8,   3,   2,   3,   4,   0,   0,   9,  11,   0,  11,   3,  -1 },
		{ 9,  11,   3,   9,   3,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   2,   7,  10,   3,   6,   0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   7,   5,   0,   7,   0,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   5,   4,   2,   7,   4,   8,   2,   4,  10,   3,   6,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   9,   2,   7,   1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   9,   2,   7,   1,   5,  11,   0,   4,   8,  -1,  -1,  -1,  -1 },
		{ 1,   0,   2,   1,   2,   7,   1,   7,  11,   3,   6,  10,  -1,  -1,  -1,  -1 },
		{ 10,   3,   6,   1,   7,  11,   1,   2,   7,   1,   4,   2,   4,   8,   2,  -1 },
		{ 9,   2,   7,   6,   4,   1,   6,   1,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   1,   3,   8,   0,   3,   6,   8,   3,   7,   9,   2,  -1,  -1,  -1,  -1 },
		{ 0,   2,   7,   0,   7,   5,   4,   1,   3,   6,   4,   3,  -1,  -1,  -1,  -1 },
		{ 2,   5,   8,   7,   5,   2,   6,   8,   5,   1,   3,   6,   5,   1,   6,  -1 },
		{ 6,   4,   5,   6,   5,  11,   6,  11,   3,   7,   9,   2,  -1,  -1,  -1,  -1 },
		{ 9,   2,   7,   0,   6,   8,   0,   3,   6,   0,   5,   3,   5,  11,   3,  -1 },
		{ 3,   4,  11,   6,   4,   3,   7,  11,   4,   0,   2,   7,   4,   0,   7,  -1 },
		{ 11,   3,   8,   8,   3,   6,   8,   2,  11,   2,   7,  11,  -1,  -1,  -1,  -1 },
		{ 9,   8,  10,   7,   9,  10,   3,   7,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   0,   7,   0,   4,   7,   4,   3,   7,   4,  10,   3,  -1,  -1,  -1,  -1 },
		{ 8,  10,   0,   0,  10,   3,   0,   3,   5,   5,   3,   7,  -1,  -1,  -1,  -1 },
		{ 10,   5,   4,   3,   5,  10,   5,   3,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   8,  10,   7,   9,  10,   3,   7,  10,   1,   5,  11,  -1,  -1,  -1,  -1 },
		{ 1,   5,  11,   9,   0,   7,   0,   4,   7,   4,   3,   7,   4,  10,   3,  -1 },
		{ 11,   0,   7,   1,   0,  11,   3,   7,   0,   8,  10,   3,   0,   8,   3,  -1 },
		{ 7,   1,   4,  11,   1,   7,   4,   3,   7,   4,  10,   3,  -1,  -1,  -1,  -1 },
		{ 4,   9,   8,   7,   9,   4,   1,   7,   4,   3,   7,   1,  -1,  -1,  -1,  -1 },
		{ 7,   1,   3,   9,   1,   7,   1,   9,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   7,   0,   0,   7,   5,   7,   8,   4,   4,   1,   3,   4,   3,   7,  -1 },
		{ 5,   1,   3,   7,   5,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   4,  11,  11,   4,   5,   4,   3,   7,   7,   9,   8,   7,   8,   4,  -1 },
		{ 3,   9,   0,   7,   9,   3,   0,  11,   3,   0,   5,  11,  -1,  -1,  -1,  -1 },
		{ 3,   7,  11,   8,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 3,   7,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,  10,  11,   7,  10,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   5,   0,   6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,   5,   8,   5,   4,   6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1 },
		{ 5,   7,   6,   5,   6,  10,   5,  10,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   7,   6,   5,   6,  10,   5,  10,   1,   4,   8,   0,  -1,  -1,  -1,  -1 },
		{ 1,   0,  10,  10,   0,   9,  10,   9,   6,   6,   9,   7,  -1,  -1,  -1,  -1 },
		{ 1,   7,  10,  10,   7,   6,   7,   1,   4,   4,   8,   9,   4,   9,   7,  -1 },
		{ 7,   6,   4,   7,   4,   1,   7,   1,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   0,   1,   8,   0,  11,   7,   8,  11,   6,   8,   7,  -1,  -1,  -1,  -1 },
		{ 7,   6,   4,   7,   4,   1,   7,   1,  11,   5,   0,   9,  -1,  -1,  -1,  -1 },
		{ 11,   6,   1,   7,   6,  11,   5,   1,   6,   8,   9,   5,   6,   8,   5,  -1 },
		{ 4,   5,   7,   4,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   7,   0,   0,   7,   8,   6,   8,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   6,   9,   9,   6,   0,   4,   0,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   9,   7,   8,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,  10,  11,   2,   8,  11,   7,   2,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,  11,   4,   4,  11,   7,   4,   7,   0,   0,   7,   2,  -1,  -1,  -1,  -1 },
		{ 8,  10,  11,   2,   8,  11,   7,   2,  11,   5,   0,   9,  -1,  -1,  -1,  -1 },
		{ 9,   4,   2,   5,   4,   9,   7,   2,   4,  10,  11,   7,   4,  10,   7,  -1 },
		{ 1,   8,  10,   2,   8,   1,   5,   2,   1,   7,   2,   5,  -1,  -1,  -1,  -1 },
		{ 1,   7,  10,   5,   7,   1,   4,  10,   7,   2,   0,   4,   7,   2,   4,  -1 },
		{ 7,   1,   9,   9,   1,   0,   1,   7,   2,   2,   8,  10,   2,  10,   1,  -1 },
		{ 7,   2,   9,  10,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   4,   2,   4,   1,   2,   1,   7,   2,   1,  11,   7,  -1,  -1,  -1,  -1 },
		{ 11,   0,   1,   7,   0,  11,   0,   7,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   0,   9,   8,   4,   2,   4,   1,   2,   1,   7,   2,   1,  11,   7,  -1 },
		{ 2,   5,   1,   9,   5,   2,   1,   7,   2,   1,  11,   7,  -1,  -1,  -1,  -1 },
		{ 4,   5,   8,   8,   5,   2,   7,   2,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   2,   0,   5,   7,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 7,   2,   4,   4,   2,   8,   4,   0,   7,   0,   9,   7,  -1,  -1,  -1,  -1 },
		{ 7,   2,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,  11,   9,   6,  10,   9,   2,   6,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,  11,   9,   6,  10,   9,   2,   6,   9,   0,   4,   8,  -1,  -1,  -1,  -1 },
		{ 5,  10,  11,   6,  10,   5,   0,   6,   5,   2,   6,   0,  -1,  -1,  -1,  -1 },
		{ 2,   5,   8,   8,   5,   4,   5,   2,   6,   6,  10,  11,   6,  11,   5,  -1 },
		{ 10,   1,   6,   1,   5,   6,   5,   2,   6,   5,   9,   2,  -1,  -1,  -1,  -1 },
		{ 0,   4,   8,  10,   1,   6,   1,   5,   6,   5,   2,   6,   5,   9,   2,  -1 },
		{ 1,   0,  10,  10,   0,   6,   2,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   6,   1,   1,   6,  10,   1,   4,   2,   4,   8,   2,  -1,  -1,  -1,  -1 },
		{ 11,   9,   1,   1,   9,   2,   1,   2,   4,   4,   2,   6,  -1,  -1,  -1,  -1 },
		{ 8,   1,   6,   0,   1,   8,   2,   6,   1,  11,   9,   2,   1,  11,   2,  -1 },
		{ 11,   6,   1,   1,   6,   4,   6,  11,   5,   5,   0,   2,   5,   2,   6,  -1 },
		{ 2,   6,   8,  11,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 6,   4,   2,   2,   4,   9,   5,   9,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   9,   6,   6,   9,   2,   6,   8,   5,   8,   0,   5,  -1,  -1,  -1,  -1 },
		{ 0,   2,   6,   0,   6,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 2,   6,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,  10,  11,   9,   8,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 0,  11,   9,   4,  11,   0,  11,   4,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,  10,  11,   0,  10,   5,  10,   0,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,  10,  11,   5,   4,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,   8,  10,   5,   8,   1,   8,   5,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   4,  10,   0,   4,   9,  10,   5,   9,  10,   1,   5,  -1,  -1,  -1,  -1 },
		{ 0,   8,  10,   1,   0,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 10,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 4,   9,   8,   1,   9,   4,   9,   1,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 1,  11,   9,   0,   1,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 11,   0,   8,   5,   0,  11,   8,   1,  11,   8,   4,   1,  -1,  -1,  -1,  -1 },
		{ 11,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 5,   9,   8,   4,   5,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 9,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ 8,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 },
		{ -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1 }
	};
	int *host_lookup = NULL;
	host_lookup = (int *)malloc(sizeof(int) * 256 * 16);

	for (int i = 0; i<256; i++) {
		for (int j = 0; j<16; j++) {
			host_lookup[i * 16 + j] = triTable[i][j];
		}
	}


	//std::cout << "\nAllocating Cubes memory\n";

	cudaStatus = hipMalloc((void **)&device_lookup, sizeof(int) * 256 * 16);
	if (cudaStatus != hipSuccess)
	{
		printf("cudamMalloc device_lookup failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_lookup, host_lookup, sizeof(int) * 256 * 16, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("cudamMemcpy device_lookup failed!\n");
		goto Error;
	}
	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	/*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

	//const int BLOCK_SIZE = 128;
	//dim3 block(BLOCK_SIZE);
	//dim3 grid(BLOCK_SIZE);
	int threadSize = 256;

	long blockSize = (sizeSolution + threadSize - 1) / threadSize;
	/*printf("blocksize:%d threadsize:%d\n", blockSize, threadSize);
	printf("gpuMatMultKernel starts\n");*/
	marching_cubes_filter << <blockSize, threadSize >> > (device_lookup, dev_mcIsLeaf, dev_mcCubeValue, dev_mcCubePosition, sizeSolution, dev_trangles, Isovalue);
	//gpuMatMultWithSharedKernelmatirx<32> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	//gpuMatMultKernel(float* source_point, float* zuobiao, int * ngbr, float* results_point, int sizeup)
	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	//printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(trangles, dev_trangles, sizeSolution * 5 * 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_mcCubeValue);
	hipFree(dev_mcCubePosition);
	hipFree(dev_trangles);
	hipFree(dev_mcIsLeaf);
	hipFree(host_lookup);


	return cudaStatus;
}